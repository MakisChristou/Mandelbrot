#include "hip/hip_runtime.h"
#include "gpu.h"

#include <stdio.h>


// Convert pixel coords to complex coords
__device__ inline long double map(long double input, long double output_start, long double output_end, long double input_start, long double input_end)
{
    return (output_start + ((output_end - output_start) / (input_end - input_start)) * (input - input_start));
}

// Stolen code from https://github.com/sevity/mandelbrot
__device__ inline Color linearInterpolation(const Color& v, const Color& u, double a)
{
	auto const b = 1 - a;

    Color color;

    color.R = b * v.R + a * u.R;
    color.G = b*v.G + a * u.G;
    color.B = b*v.B + a * u.B;

	return color;
}

__device__ inline cIterations iterateMandelbrot(long double i, long double j, long double output_start, long double output_end, int image_width, int image_height, int n_max)
{

    // printf("i: %lf\n", i);
    // printf("j: %lf\n", j);
    

    long double complex_i = map(i, output_start, output_end, 0, image_width);
    long double complex_j = map(j, output_start, output_end, 0, image_height);

    

    long double x0 = 0;
    long double y0 = 0;
    
    long double x = 0;
    long double y = 0;

    int n = 0; // iterations

    while(x*x + y*y <= 4 && n < n_max)
    {
        x = x0*x0 - y0*y0 + complex_i;
        y = 2*x0*y0 + complex_j;

        x0 = x;
        y0 = y;
        n++;
    }

    // printf("complex_i: %lf\n", complex_i);
    // printf("complex_j: %lf\n", complex_j);
    

    cIterations citerations;
    citerations.n = n;
    citerations.c.Re = x;
    citerations.c.Re = y;

    return citerations;
}

__device__ inline Color getColor(int iter, Color* colorPallete, int palleteSize, int n_max)
{
    // Stolen Code from https://github.com/sevity/mandelbrot
    

    size_t max_color = palleteSize - 1;

    // if (iter == n_max)iter = 0;
    double mu = 1.0*iter / n_max;


    //scale mu to be in the range of colors
    mu *= max_color;
    size_t i_mu = static_cast<size_t>(mu);

    Color color1 = colorPallete[i_mu];
    Color color2;

    

    if(i_mu + 1 < max_color)
    {
        color2 = colorPallete[i_mu + 1];
    }
    else
    {
        color2 = colorPallete[max_color];
    }

    Color c = linearInterpolation(color1, color2, mu - i_mu);

    if(iter == n_max)
    {
        c.R = c.G = c.B = 0;
    }

    return c;
}

// Kernel
__global__ void mandelbortKernel(Color *pixelColours, Color* colorPallete, int palleteSize, int image_height, int image_width, double* output_start, double* output_end, int n_max, int s_max)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; 
    int j = blockIdx.y * blockDim.y + threadIdx.y;


    if (i < image_width && j < image_height)
    {
        cIterations citerations;
        Complex c;
        int n = 0;
        int sum = 0;


        for(double k = 0.0; k < 1.0; k+=1.0/s_max)
        {
            double ii  = i+k;
            double jj = j+k;

            // Pass value of bounds
            citerations = iterateMandelbrot(ii,jj, *output_start, *output_end, image_width, image_height, n_max);

            n = citerations.n;
            c = citerations.c;
            
            sum+=n;
        }

        sum = sum / s_max;
        n = sum;
        
        // bug here, cannot access colorPallete (and probably not other arrays either)
        Color color = getColor(n, colorPallete, palleteSize, n_max);

        // printf("%d%d%d\n",color.R,color.G,color.B);

        // printf("*output_start: %lf\n", *output_start);
        // printf("*output_end: %lf\n", *output_end);
        // printf("n_max: %d\n", n_max);
        // printf("s_max: %d\n", s_max);
        // printf("image_width: %d\n", image_width);
        // printf("image_height: %d\n", image_height);

        // printf("Hello1 thread row=%d, col=%d, n = %d \n", i, j, n);

        // iterationCounts[i * image_height + j] = n;
        pixelColours[i * image_height + j] = color;
    } 
}


Color* gpuAllocColor(int N, int M, int bytes)
{
    Color *d_B;
    hipMalloc(&d_B, bytes);
    return d_B;
}

double* gpuAllocDouble(int N, int M, int bytes)
{
    double* d_output_start;
    hipMalloc(&d_output_start, bytes);
    return d_output_start;
}

void gpuFree(Color* d_B, Color* d_P, double* d_output_start, double* d_output_end)
{
        hipFree(d_B);
        hipFree(d_P);
        hipFree(d_output_start);
        hipFree(d_output_end);
}

void gpuCopyToDevice(int N, int M, int palleteSize, Color* d_B, Color* d_P, Color* B, Color* P, double* d_output_start, double* d_output_end, double* output_start_host, double* output_end_host)
{
    size_t pixelBytes = N*M*sizeof(Color);
    size_t palleteBytes = palleteSize*sizeof(Color);

    hipMemcpy(d_B, B, pixelBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_P, P, palleteBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_output_start, output_start_host, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_output_end, output_end_host, sizeof(double), hipMemcpyHostToDevice);

}

void gpuCopyFromDevice(int N, int M, int palleteSize, Color* d_B, Color* d_P, Color* B, Color* P)
{   
    size_t pixelBytes = N*M*sizeof(Color);
    size_t palleteBytes = palleteSize*sizeof(Color);


    hipMemcpy(B, d_B, pixelBytes, hipMemcpyDeviceToHost);
}

void gpuRender(Color* d_B, Color* d_P, int palleteSize, int N, int M, double* d_output_start, double* d_output_end, int n_max, int s_max)
{
    int thr_per_blk = 16;
    int blk_in_grid = (N + thr_per_blk -1 )/ thr_per_blk ;

    dim3 threads(thr_per_blk, thr_per_blk);
    dim3 blocks(blk_in_grid, blk_in_grid);

    mandelbortKernel<<< blocks, threads >>>(d_B, d_P, palleteSize, N, M, d_output_start, d_output_end, n_max, s_max);

    hipDeviceSynchronize();
}