#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cstdlib>
#include <math.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <chrono>


using namespace std::chrono;

// #include <SDL2/SDL.h>

// Global Declarations
int image_width = 5000;
int image_height = 5000; 



// long double output_start = 0.2f;
// long double output_end = 0.5f;

// long double output_start = 0.35f;
// long double output_end = 0.36f;

int n_max = 64; // 4096
int s_max = 8; // prefer to be a power of 2

typedef struct gpuColor
{
    int R;
    int G;
    int B;

    __device__ __host__ gpuColor()
    {

    }

    __device__ __host__ gpuColor(int r, int g, int b)
    {
        R = r;
        G = g;
        B = b;
    }
    __device__ __host__ gpuColor(double r, double g, double b)
    {
        R = (int)r;
        G = (int)g;
        B = (int)b;
    }
};

typedef struct Color
{
    int R;
    int G;
    int B;

    Color()
    {
        R = 0;
        G = 0;
        B = 0;
    }

    Color(int r, int g, int b)
    {
        R = r;
        G = g;
        B = b;
    }
};

typedef struct Complex
{
    long double Re;
    long double Im;
};

typedef struct cIterations
{
    Complex c;
    int n;
};

// Convert pixel coords to complex coords
__device__ inline long double map(long double input, long double output_start, long double output_end, long double input_start, long double input_end)
{
    return (output_start + ((output_end - output_start) / (input_end - input_start)) * (input - input_start));
}

// Stolen code from https://github.com/sevity/mandelbrot
__device__ inline gpuColor linearInterpolation(const gpuColor& v, const gpuColor& u, double a)
{
	auto const b = 1 - a;
	return gpuColor(b * v.R + a * u.R, b*v.G + a * u.G, b*v.B + a * u.B);
}

__device__ inline cIterations iterateMandelbrot(long double i, long double j, long double output_start, long double output_end, int image_width, int image_height, int n_max)
{

    // printf("i: %lf\n", i);
    // printf("j: %lf\n", j);
    

    long double complex_i = map(i, output_start, output_end, 0, image_width);
    long double complex_j = map(j, output_start, output_end, 0, image_height);

    

    long double x0 = 0;
    long double y0 = 0;
    
    long double x = 0;
    long double y = 0;

    int n = 0; // iterations

    while(x*x + y*y <= 4 && n < n_max)
    {
        x = x0*x0 - y0*y0 + complex_i;
        y = 2*x0*y0 + complex_j;

        x0 = x;
        y0 = y;
        n++;
    }

    // printf("complex_i: %lf\n", complex_i);
    // printf("complex_j: %lf\n", complex_j);
    

    cIterations citerations;
    citerations.n = n;
    citerations.c.Re = x;
    citerations.c.Re = y;

    return citerations;
}

__device__ inline gpuColor getColor(int iter, gpuColor* colorPallete, int palleteSize, int n_max)
{
    // Stolen Code from https://github.com/sevity/mandelbrot
    

    size_t max_color = palleteSize - 1;

    // if (iter == n_max)iter = 0;
    double mu = 1.0*iter / n_max;


    //scale mu to be in the range of colors
    mu *= max_color;
    size_t i_mu = static_cast<size_t>(mu);

    gpuColor color1 = colorPallete[i_mu];
    gpuColor color2;

    

    if(i_mu + 1 < max_color)
    {
        color2 = colorPallete[i_mu + 1];
    }
    else
    {
        color2 = colorPallete[max_color];
    }

    gpuColor c = linearInterpolation(color1, color2, mu - i_mu);

    if(iter == n_max)
    {
        c.R = c.G = c.B = 0;
    }

    return c;
}

// Kernel
__global__ void mandelbortKernel(gpuColor *pixelColours, gpuColor* colorPallete, int palleteSize, int image_height, int image_width, double* output_start, double* output_end, int n_max, int s_max)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; 
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // For some reason
    // double output_start = *output_startt;
    // double output_end = *output_endd;
    // double factor = *factorr;


    if (i < image_width && j < image_height)
    {
        cIterations citerations;
        Complex c;
        int n = 0;
        int sum = 0;


        for(double k = 0.0; k < 1.0; k+=1.0/s_max)
        {
            double ii  = i+k;
            double jj = j+k;

            citerations = iterateMandelbrot(ii,jj, *output_start, *output_end, image_width, image_height, n_max);

            n = citerations.n;
            c = citerations.c;
            
            sum+=n;
        }

        sum = sum / s_max;
        n = sum;
        
        // bug here, cannot access colorPallete (and probably not other arrays either)
        gpuColor color = getColor(n, colorPallete, palleteSize, n_max);

        // printf("%d%d%d\n",color.R,color.G,color.B);

        // printf("*output_start: %lf\n", *output_start);
        // printf("*output_end: %lf\n", *output_end);
        // printf("n_max: %d\n", n_max);
        // printf("s_max: %d\n", s_max);
        // printf("image_width: %d\n", image_width);
        // printf("image_height: %d\n", image_height);

        // printf("Hello1 thread row=%d, col=%d, n = %d \n", i, j, n);

        // iterationCounts[i * image_height + j] = n;
        pixelColours[i * image_height + j] = color;
    } 
}


// Returns smooth colour based on iteration and C value when escape
inline long double smoothColor(int n, Complex c)
{
    long double Zr = c.Re;
    long double Zi = c.Im; 
    long double N = (long double) n;

    // std::cerr << "Zr: " << Zr << " Zi: " << Zi << " N: " << N << std::endl;

    return 1.0 + N - log(log(sqrt(Zr*Zr + Zi*Zi)));
}

// Prints PPM in std
void writePPM(gpuColor* pixelColors)
{
    printf("P3\n");
    printf("%d %d",image_width,image_height);
    printf("\n255\n");

    for(int j = image_height-1; j >=0; --j)
    {
        for(int i = 0; i < image_width; ++i)
        {
            gpuColor c = pixelColors[i*image_height + j];
            printf("%d %d %d\n", c.R, c.G, c.B);
        }
    }
}


inline std::vector<Color> generateColorPalete()
{
    std::vector<Color> colorPalete;

    Color color1;
    color1.R = 0xFF;
    color1.G = 0xFF;
    color1.B = 0xFF;

    Color color2;
    color2.R = 0x00;
    color2.G = 0x00;
    color2.B = 0x00;

    Color black;
    black.R = 0x00;
    black.G = 0x00;
    black.B = 0x00;

    long double sumR = 0;
    long double sumG = 0;
    long double sumB = 0;



    for(int i = 0; i < n_max; i++)
    {
        if(i == 0)
        {
            colorPalete.push_back(color1);
        }
        else if(i == n_max-1)
        {
            colorPalete.push_back(color2);
        }
        else if(i == n_max)
        {
            colorPalete.push_back(black);
        }
        else
        {
            Color tempColor;

            // Calculate log step for current iteration
            double long rStep = log2(1 + (color1.R * log2(i)) / 10) / 8;
            double long gStep = log2(1 + (color1.G * log2(i)) / 10) / 8;
            double long bStep = log2(1 + (color1.B * log2(i)) / 10) / 8;

            // 
            sumR += rStep;
            sumG += gStep;
            sumB += bStep;

            tempColor.R = sumR;
            tempColor.G = sumG;
            tempColor.B = sumB;

            colorPalete.push_back(tempColor);
        }
    }

    return colorPalete;
}


int main(int argc, char* argv[])
{
    // SDL_Init(SDL_INIT_EVERYTHING);
    // SDL_Window* window;
    // SDL_Renderer* renderer;
    // SDL_Event event;
    // SDL_CreateWindowAndRenderer(image_width, image_height, 0, &window, &renderer);
    // SDL_RenderSetLogicalSize(renderer, image_width, image_height);


    int N = image_width;
    int M = image_height; 

    double output_start = 0.35f;
    double output_end = 0.36f;
    double factor = 1.0f;

     int palleteSize = 5;

    int *iterationCounts = (int*) malloc(N*M*sizeof(int));
    Color *pixelColours = (Color*) malloc(N*M*sizeof(Color));

    while(1)
    {
        // SDL_RenderPresent(renderer);
        // SDL_Event event;
        // while (SDL_PollEvent(&event))
        // {
        //     if (event.type == SDL_QUIT) {
        //         SDL_Quit();
        //         return 0;
        //     }
        // }
    



        // Number of bytes to allocate for N doubles
        // size_t iterationBytes = N*M*sizeof(int);
        size_t pixelBytes = N*M*sizeof(gpuColor);
        size_t palleteBytes = palleteSize*sizeof(gpuColor);

        // Allocate memory for arrays A, B, and C on host
        // int *A = (int*)malloc(iterationBytes); // iterationCounts
        gpuColor *B = (gpuColor*)malloc(pixelBytes); // pixelColours
        gpuColor *P = (gpuColor*)malloc(palleteBytes); // colorPalete
        double* output_start_host = (double*)malloc(sizeof(double));
        double* output_end_host = (double*)malloc(sizeof(double));
        double* factor_host = (double*)malloc(sizeof(double));


        // Allocate memory for arrays d_A, d_B, and d_C on device
        int *d_A;
        gpuColor *d_B;
        gpuColor *d_P;
        double* d_output_start;
        double* d_output_end;

        // hipMalloc(&d_A, iterationBytes);
        hipMalloc(&d_B, pixelBytes);
        hipMalloc(&d_P, palleteBytes);
        hipMalloc(&d_output_start, sizeof(double));
        hipMalloc(&d_output_end, sizeof(double));

        // Fill host arrays data structures
        //  for(int i = 0; i < image_width; i++)
        // {
        //     for(int j = 0; j < image_height; j++)
        //     {
        //         B[i*image_height+j].R = 0;
        //         B[i*image_height+j].G = 0;
        //         B[i*image_height+j].B = 0;
        //         // A[i*image_height+j] = 0;
        //     }
        // }

        P[0] = gpuColor(0,7,100);
        P[1] = gpuColor(32,107,203);
        P[2] = gpuColor(237,255,255);
        P[3] = gpuColor(255,170,0);
        P[4] = gpuColor(0,2,0);

        // Fill host arrays data structures
        output_start_host[0] = output_start;
        output_end_host[0] = output_end;


        // Copy data from host arrays A and B to device arrays d_A and d_B
        // hipMemcpy(d_A, A, iterationBytes, hipMemcpyHostToDevice);
        hipMemcpy(d_B, B, pixelBytes, hipMemcpyHostToDevice);
        hipMemcpy(d_P, P, palleteBytes, hipMemcpyHostToDevice);
        hipMemcpy(d_output_start, output_start_host, sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_output_end, output_end_host, sizeof(double), hipMemcpyHostToDevice);
        

        // Set execution configuration parameters
        //      thr_per_blk: number of CUDA threads per grid block
        //      blk_in_grid: number of blocks in grid
        int thr_per_blk = 16;
        int blk_in_grid = (N + thr_per_blk -1 )/ thr_per_blk ;

        dim3 threads(thr_per_blk, thr_per_blk);
        dim3 blocks(blk_in_grid, blk_in_grid);

        auto start = high_resolution_clock::now();

        mandelbortKernel<<< blocks, threads >>>(d_B, d_P, palleteSize, N, N, d_output_start, d_output_end, n_max, s_max);

        hipDeviceSynchronize();

        auto stop = high_resolution_clock::now();
        auto duration = duration_cast<microseconds>(stop - start);


        std::cerr << "Mandel frame computed in: " << duration.count() << "us" << "\n";

        // Copy data from device array d_C to host array C
        hipMemcpy(B, d_B, pixelBytes, hipMemcpyDeviceToHost);


        writePPM(B);

        // Free CPU memory
        // free(A);
        free(B);
        free(P);
        free(output_start_host);
        free(output_end_host);

        // Free GPU memory
        // hipFree(d_A);
        hipFree(d_B);
        hipFree(d_P);
        hipFree(d_output_start);
        hipFree(d_output_end);

        // printf("\n---------------------------\n");
        // printf("__SUCCESS__\n");
        // printf("---------------------------\n");
        // printf("N                 = %d\n", K);
        // printf("Threads Per Block = %d\n", thr_per_blk);
        // printf("Blocks In Grid    = %d\n", blk_in_grid);
        // printf("---------------------------\n\n");


        // printf("Rendered! \n");

        return 0;



        // // Main Loop
        // for(int i = 0; i < image_width; i++)
        // {
        //     // std::cerr << "\rScanlines remaining: " << image_width - i << " " <<  std::flush;

        //     for(int j = 0; j < image_height; j++)
        //     {

        //         cIterations citerations;
        //         Complex c;
        //         int n = 0;
        //         int sum = 0;
  


        //         for(double k = 0.0; k < 1.0; k+=1.0/s_max)
        //         {
        //             double ii  = i+k;
        //             double jj = j+k;

        //             citerations = iterateMandelbrot(ii,jj);

        //             n = citerations.n;
        //             c = citerations.c;
                    
        //             sum+=n;
        //         }

        //         sum = sum / s_max;
        //         n = sum;

        //         Color color = getColor(n, colorPallete, palleteSize);

        //         iterationCounts[i * image_height + j] = n;
        //         pixelColours[i * image_height + j] = color;
                

        //         // SDL Draw
        //         // SDL_SetRenderDrawColor(renderer, color.R, color.G, color.B, 255);
        //         // SDL_RenderDrawPoint(renderer, i, j);
        //     }
        // }

        // Zoom in code by https://www.youtube.com/watch?v=KnCNfBb2ODQ
        // output_start+=0.15*factor;
        // output_end-=0.1*factor;
        // factor *= 0.9349;
        // n_max+=5;

        

        return 0;

    }

    free(iterationCounts);
    free(pixelColours);

    return 0;
}